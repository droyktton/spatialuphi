#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include<thrust/reduce.h>
#include<fstream>
#include<cstdlib>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <hipfft/hipfft.h>
#include "cutil.h"
#include <chrono>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>


// harmonic elasticity constant
#ifndef C2
#define C2 1.0   
#endif

// anharmonic elasticity constant
#ifndef C4
#define C4 0.0   
#endif

#ifndef TEMP
#define TEMP 0.1
#endif

/*#ifndef seed    
#define seed 1234
#endif
*/

#ifndef TAU    
#define TAU 0.1
#endif


#ifndef TILT
#define TILT 0.0
#endif

#ifndef MONITOR
#define MONITOR 1000
#endif

#ifndef MONITORCONF
#define MONITORCONF 100000
#endif

#ifdef DOUBLE
typedef double real;
typedef hipfftDoubleComplex complex;
#else
typedef float real;
typedef hipfftComplex complex;
#endif

// file to log parameters of the run
std::ofstream logout("logfile.dat");


// main class:
class cuerda{

    public:
    cuerda(unsigned long _L, real _dt, unsigned long _seed):L(_L),dt(_dt),fourierCount(0),seed(_seed)
    {
        // interface position
        u.resize(L);
        
        // interface forces
        force_u.resize(L);
 
        // interface forces
        noise.resize(L);
        thrust::fill(noise.begin(),noise.end(),real(0.0));
        warmup_noise(); // warmup noise
 
        // flat initial condition
        thrust::fill(u.begin(),u.end(),real(0.0));
        
        // plans for the interface structure factor
        #ifdef DOUBLE
        CUFFT_SAFE_CALL(hipfftPlan1d(&plan_r2c,L,HIPFFT_D2Z,1));
        #else
        CUFFT_SAFE_CALL(hipfftPlan1d(&plan_r2c,L,HIPFFT_R2C,1));
        #endif

	    int Lcomp=L/2+1;
	    Fou_u.resize(Lcomp); // interface position in fourier space

        acum_Sofq_u.resize(L); // average structure factor
        inst_Sofq_u.resize(L); // instantaneous structure factor

        // initialization of structure factors   
        thrust::fill(acum_Sofq_u.begin(),acum_Sofq_u.end(),real(0.0));

        #ifdef DEBUG
        std::cout << "L=" << L << ", dt=" << dt << std::endl;
        #endif
    }

    void flat_initial_condition(){
        // flat initial condition
        thrust::fill(u.begin(),u.end(),real(0.0));
    }

    void warmup_noise(){
        real dt_ = dt;
        unsigned long seed_ = seed;
        unsigned long L_ = L;
        
        unsigned long twarm = (unsigned long )(100.*TAU/dt_); // number of warmup steps
        for(unsigned long n=0;n<twarm;n++)
        {
            thrust::for_each(
                thrust::make_zip_iterator(
                    thrust::make_tuple(noise.begin(),thrust::make_counting_iterator((unsigned long)0))        
                ),
                thrust::make_zip_iterator(
                    thrust::make_tuple(noise.end(),thrust::make_counting_iterator((unsigned long)L_))        
                ),
                [=] __device__ (thrust::tuple<real &,unsigned long> t)
                {
                    unsigned long i=thrust::get<1>(t);
                    hiprandStatePhilox4_32_10_t state;
                    hiprand_init(seed_, i, n, &state);
                    float ran = sqrt(2*TEMP*dt_)*hiprand_normal(&state);
                    thrust::get<0>(t) += -thrust::get<0>(t)*dt_/TAU + ran;
                } 
            );  
        }
    };


    // reset structure factor acumulator
    void reset_acum_Sofq(){
        thrust::fill(acum_Sofq_u.begin(),acum_Sofq_u.end(),real(0.0));
    }

    // returns the center of mass position
    real center_of_mass()
    {
        //DANGER: large sum over large numbers
        real cmu = thrust::reduce(u.begin(),u.end(),real(0.0))/L;
        return cmu;
    }

    // returns the center of mass velocity
    real center_of_mass_velocity()
    {
        //SAFE: velocities are bounded
        real vcmu = thrust::reduce(force_u.begin(),force_u.end(),real(0.0))/L;
        return vcmu;
    }

    // computes the instantaneous and acumulated structure factor
    void fourier_transform(){

        real *raw_u = thrust::raw_pointer_cast(&u[0]); 
        complex *raw_fou_u = thrust::raw_pointer_cast(&Fou_u[0]); 

        // raw_u --> transform --> raw_fou_u
        #ifdef DOUBLE
        CUFFT_SAFE_CALL(hipfftExecD2Z(plan_r2c, raw_u, raw_fou_u));
        #else
	    CUFFT_SAFE_CALL(hipfftExecR2C(plan_r2c, raw_u, raw_fou_u));
        #endif

        // compute the structure factor from fourier components
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(Fou_u.begin(),acum_Sofq_u.begin(),inst_Sofq_u.begin())
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(Fou_u.end(),acum_Sofq_u.end(),inst_Sofq_u.end())
            ),
            [=] __device__ (thrust::tuple<complex,real &,real &> t)
            {
                complex fu=thrust::get<0>(t);
                real sofq = fu.x*fu.x + fu.y*fu.y;
                thrust::get<1>(t) += sofq; // average structure factor 
                thrust::get<2>(t) = sofq; //instantaneous structure factor
            }
        );
        fourierCount++; // increment the number of fourier transforms
    }

    // computes the center of mass, the variance (roughness)
    // and the leading and receding points of the interface 
    thrust::tuple<real, real, real, real> roughness()
    {
        // CHECK for large numbers!
        
        // center of mass displacement
        real cmu = thrust::reduce(u.begin(),u.end(),real(0.f),thrust::plus<real>())/real(L);
	    
	    // extreme displacements
	    real u0=u[0]; 
        real maxu = thrust::reduce(u.begin(),u.end(),u0,thrust::maximum<real>());
        real minu = thrust::reduce(u.begin(),u.end(),u0,thrust::minimum<real>());

        // variance or roughness
        real cmu2 = 
        thrust::transform_reduce(
            u.begin(),u.end(),
            [=] __device__ (real x){
                return (x-cmu)*(x-cmu);
            },
            real(0.f),
            thrust::plus<real>()
        )/real(L);

        return thrust::make_tuple(cmu,cmu2,maxu,minu);
    }

    // just compute and prints center of mass in out stream
    void print_center_of_mass(std::ofstream &out)
    {
        real cm=center_of_mass();    
        out << cm << std::endl;
    }

    // rescale all position in order to avoid large displacements
    void rescale()
    {
        real cmu=center_of_mass();

        thrust::transform(u.begin(),u.end(),u.begin(),
        [=] __device__ (real u){
            return u-cmu;
        }
        );
    };

    // print roughness results
    void print_roughness(std::ofstream &out, real t)
    {
        real vcm=center_of_mass_velocity();

        thrust::tuple<real,real,real,real> cm = roughness();

        //get cmu,cmu2,maxu,minu
        real cmu = thrust::get<0>(cm);
        real cmu2 = thrust::get<1>(cm);
        real maxu = thrust::get<2>(cm);
        real minu = thrust::get<3>(cm);

        out << t << " " << vcm << " " << cmu << " " << " " << cmu2 << " " << maxu << " " << minu << std::endl;
    }

    // Computes the forces and advance one time step using Euler method
    void update(unsigned long n)
    {
        real *raw_u = thrust::raw_pointer_cast(&u[0]); 
        real *raw_noise = thrust::raw_pointer_cast(&noise[0]); 

        // variables to be camptured by lambda (not elegant...)
        real dt_=dt;
        unsigned long L_ = L;
        unsigned long seed_ = seed;
        // Forces
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(force_u.begin(),thrust::make_counting_iterator((unsigned long)0))        
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(force_u.end(),thrust::make_counting_iterator((unsigned long)L))        
            ),
            [=] __device__ (thrust::tuple<real &,unsigned long> t)
            {
                unsigned long i=thrust::get<1>(t);
                unsigned long ileft = (i-1+L_)%L_;
                unsigned long iright = (i+1)%L_;

                real uleft = raw_u[ileft];
                real uright = raw_u[iright];
                
                // optional to impose tilted boundary conditions
                #ifdef TILT
                if(i==0) {
                    uleft -= L_*TILT;
                }  
                if(i==L_-1){
                    uright += L_*TILT;
                }  
                #endif
                
                // correlated noise update 
                hiprandStatePhilox4_32_10_t state;
                hiprand_init(seed_, i, n, &state);
                float ran = sqrt(2*TEMP*dt_)*hiprand_normal(&state);
                raw_noise[i] += -raw_noise[i]*dt_/TAU + ran;
                                        
                real lap_u = C2*(uright + uleft - 2.0*raw_u[i]) + raw_noise[i];
                
                // modify element force
                thrust::get<0>(t) = C2*lap_u;
            } 
        );

        #ifdef DEBUG
        std::cout << "updating" << std::endl;
        #endif

        // Euler step: u = u + dt*force_u
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(u.begin(), force_u.begin())        
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(u.end(),force_u.end())        
            ),
            [=] __device__ (thrust::tuple<real &,real> t)
            {
                thrust::get<0>(t) = thrust::get<0>(t) + dt_*thrust::get<1>(t);
            } 
        );
    };

    // print the whole configuration to a file
    void print_config(std::ofstream &out){
        real cm = center_of_mass();

        for(int i=0;i<L;i++){
            out << u[i] << " " << cm << "\n";
        }
        out << "\n" << std::endl;
    };

    // prints the whole averaged structure factor to a file
    void print_sofq(std::ofstream &out){
        for(int i=0;i<L;i++){
            out << acum_Sofq_u[i]/fourierCount << "\n";
        }
        out << "\n" << std::endl;
    };

    // prints the instantaneous structure factor to a file
    void print_inst_sofq(std::ofstream &out){
        for(int i=0;i<L;i++){
            out << inst_Sofq_u[i] << "\n";
        }
        out << "\n" << std::endl;
    };

    // variables and arrays of the class
    private:
        real dt;
        unsigned long L;
        unsigned long seed;
        
        real f0;
        thrust::device_vector<real> u;
        thrust::device_vector<real> force_u;

        thrust::device_vector<real> noise;

        int fourierCount;
        hipfftHandle plan_r2c;
        thrust::device_vector<complex> Fou_u;

        thrust::device_vector<real> acum_Sofq_u;
	    thrust::device_vector<real> inst_Sofq_u;
	    
};

int main(int argc, char **argv){
    // Get the current CUDA device
    int device;
    hipGetDevice(&device);

    // Get the properties of the current CUDA device
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    std::ofstream confout("conf.dat");
    confout << "#u[i]" << " " << "cmu" << "\n";

    std::ofstream sofqout("sofq.dat");
    sofqout << "#av_Sofq_u[i]" << "\n";

    std::ofstream instsofqout("inst_sofq.dat");
    instsofqout << "#inst_Sofq_u[i]" << "\n";

    std::ofstream cmout("cm.dat");
    cmout << "#t" << " " << "velu" << " " << "cmu" << " " << "cmu2" << " " << "maxu" << " " << "minu" << std::endl;

    std::ofstream lastconfout("lastconf.dat");
    lastconfout << "#u[i]" << " " << "cmu" << "\n";

    if(argc!=4){
        std::cout << "Usage: " << argv[0] << " L Nrun seed" << std::endl;
        std::cout << "L: interface length" << std::endl;
        std::cout << "Nrun: number of running steps" << std::endl;
        std::cout << "seed: random seed" << std::endl;
        return 1;
    }

    unsigned int L=atoi(argv[1]); //interface lenght
    unsigned long Nrun = atoi(argv[2]); // running steps
    unsigned long seed = atoi(argv[3]); // global seed
    
    // time step
    real dt=0.1;

    // equilibration
    unsigned long Neq = int(Nrun*0.75); // number of equilibration steps

    // instance
    cuerda C(L,dt,seed);

    #ifdef DOUBLE
    logout << "double precision\n";
    #else
    logout << "simple precision\n";
    #endif
    #ifdef TILT
    logout << "TILT= " << TILT << "\n";
    #endif
    #ifdef TEMP
    logout << "TEMP= " << TEMP << "\n";
    #endif    
    #ifdef seed
    logout << "seed= " << seed << "\n";
    #endif 
    #ifdef MONITOR
    logout << "MONITOR= " << MONITOR << "\n";
    #endif
    #ifdef MONITORCONF
    logout << "MONITORCONF= " << MONITORCONF << "\n";
    #endif
    
    
    logout 
	<< "C2= " << C2  << "\n"
	<< "C4= " << C4 << "\n"
	<< ", dt= " << dt << "\n"
	<< ", L= " << L << std::endl;
    logout.flush();

    // Start the timer
    auto start = std::chrono::high_resolution_clock::now();

    unsigned long jlog=1;

    for(int i=0;i<Nrun;i++){
        C.update(i);

        if(i%MONITORCONF==0){
            C.print_config(confout);
            C.fourier_transform();
            C.print_inst_sofq(instsofqout);
        }
                
        // print configs and structure factors at 1,10,100,etc...        
        if(i%jlog==0){
            C.print_config(confout);
            C.fourier_transform();
            C.print_inst_sofq(instsofqout);
            jlog*=10;
        }
        
        if(i%Neq==0) C.reset_acum_Sofq();
                        
        if(i%MONITOR==0){
            C.print_roughness(cmout,dt*i);
        }
    }

    // Stop the timer
    auto end = std::chrono::high_resolution_clock::now();

    C.print_config(confout);
    C.print_sofq(sofqout);

    // Calculate the duration
    std::chrono::duration<double> duration = end - start;
    // Output the duration
       
    logout << ", Time taken: " << duration.count() << " seconds"
    << ", device= " << deviceProp.name << std::endl;

    return 0;
}

/*
nvcc --expt-extended-lambda -lcufft main.cu -DCu=0.0 -DCphi=0.0 -DEpsilon=0.001 -std=c++14 -arch=sm_61 -o a0.out
*/
